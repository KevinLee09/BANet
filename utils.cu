#include "hip/hip_runtime.h"
//
//  lmbspecialops - a collection of tensorflow ops
//  Copyright (C) 2017  Benjamin Ummenhofer, Huizhong Zhou
//  
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//  
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//  
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <http://www.gnu.org/licenses/>.
//
#define EIGEN_USE_GPU
#include "/usr/local/cuda/include/hip/hip_vector_types.h"
#include "/usr/local/cuda/include/hip/hip_runtime.h"
#include "tensorflow/core/framework/op_kernel.h"
#include "tensorflow/core/platform/logging.h"
#include "tensorflow/core/platform/platform.h"

#include "tensorflow/stream_executor/cuda/cuda_platform_id.h"
#include "tensorflow/stream_executor/device_memory.h"
#include "tensorflow/stream_executor/dnn.h"
#include "tensorflow/stream_executor/event.h"
#include "tensorflow/stream_executor/host/host_platform_id.h"
#include "tensorflow/stream_executor/lib/status.h"
#include "tensorflow/stream_executor/multi_platform_manager.h"
#include "tensorflow/stream_executor/platform.h"
#include "tensorflow/stream_executor/scratch_allocator.h"
#include "tensorflow/stream_executor/stream.h"
#include "tensorflow/stream_executor/kernel.h"
#include "tensorflow/stream_executor/stream_executor.h"

#include "tensorflow/core/platform/types.h"
#include "tensorflow/core/framework/shape_inference.h"
#include "tensorflow/core/framework/register_types.h"
#include "unsupported/Eigen/CXX11/src/Tensor/TensorDeviceCuda.h"


using namespace tensorflow;
inline int divup(int x,int y){
  div_t tmp = std::div(x,y);
  return tmp.quot+(tmp.rem!=0?1:0);
}
struct CudaLaunchConfig {
  int virtual_thread_count = -1;
  int thread_per_block = -1;
  int block_count = -1;
};

const int d_getNumGpuMultiProcessors=24;
const int d_maxGpuThreadsPerMultiProcessor=2048;
const int d_maxGpuThreadsPerBlock=1024;

inline CudaLaunchConfig GetCudaLaunchConfig(int work_element_count){
  CHECK_GT(work_element_count, 0);
  CudaLaunchConfig config;
  const int virtual_thread_count = work_element_count;
  const int physical_thread_count = std::min(
      d_getNumGpuMultiProcessors*d_maxGpuThreadsPerMultiProcessor,
      virtual_thread_count);
  const int thread_per_block = std::min(1024,d_maxGpuThreadsPerBlock);
  const int block_count =std::min(divup(physical_thread_count,thread_per_block),d_getNumGpuMultiProcessors);

  config.virtual_thread_count = virtual_thread_count;
  config.thread_per_block = thread_per_block;
  config.block_count = block_count;
  return config;
}



#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

inline const hipStream_t& GetCudaStream(OpKernelContext* context) {
  const hipStream_t* ptr = CHECK_NOTNULL(
      reinterpret_cast<const hipStream_t*>(context->op_device_context()
                                                ->stream()
                                                ->implementation()
                                                ->GpuStreamMemberHack()));
  return *ptr;
}

perftools::gputools::DeviceMemory<float> AsDeviceMemory(const float* cuda_memory) {
  perftools::gputools::DeviceMemoryBase wrapped(const_cast<float*>(cuda_memory));
  perftools::gputools::DeviceMemory<float> typed(wrapped);
  return typed;
}

class CublasScratchAllocator : public perftools::gputools::ScratchAllocator {
 public:
  using Stream = ::perftools::gputools::Stream;
  using DeviceMemoryBytes = ::perftools::gputools::DeviceMemory<uint8>;

  CublasScratchAllocator(OpKernelContext* context) : context_(context) {}

  int64 GetMemoryLimitInBytes(Stream* stream) override { return -1; }

  perftools::gputools::port::StatusOr<DeviceMemoryBytes> AllocateBytes(
      Stream* stream, int64 byte_size) override {
    Tensor temporary_memory;

    Status allocation_status(context_->allocate_temp(
        DT_UINT8, TensorShape({byte_size}), &temporary_memory));
    if (!allocation_status.ok()) {
      return perftools::gputools::port::StatusOr<DeviceMemoryBytes>(
          DeviceMemoryBytes::MakeFromByteSize(nullptr, 0));
    }
    // Hold the reference of the allocated tensors until the end of the
    // allocator.
    allocated_tensors_.push_back(temporary_memory);
    return perftools::gputools::port::StatusOr<DeviceMemoryBytes>(
        DeviceMemoryBytes::MakeFromByteSize(
            temporary_memory.flat<uint8>().data(),
            temporary_memory.flat<uint8>().size()));
  }

 private:
  OpKernelContext* context_;
  std::vector<Tensor> allocated_tensors_;
};


REGISTER_OP("WarpComputation")
    .Input("imgs_flat:float")
    .Input("index:int32")
    .Input("w:float")
    .Output("output:float");

__global__ void warp_kernel(float* out, const float* in, const int* index,const float* weights,const int npixels,const int nchannels){
  int pixel   = blockIdx.x * blockDim.x + threadIdx.x;
  int channel = blockIdx.y * blockDim.y + threadIdx.y;
  if( channel >= nchannels||pixel >= npixels)
    return;
  out[nchannels*pixel+channel]=in[nchannels*index[4*pixel]+channel]*weights[4*pixel]
                              +in[nchannels*index[4*pixel+1]+channel]*weights[4*pixel+1]
                              +in[nchannels*index[4*pixel+2]+channel]*weights[4*pixel+2]
                              +in[nchannels*index[4*pixel+3]+channel]*weights[4*pixel+3];
}



REGISTER_OP("EquationConstruction")
     .Input("jacobian:float")
     .Input("gradient:float")
     .Input("difference:float")
     .Output("left:float")
     .Output("right:float")
     .SetShapeFn([](shape_inference::InferenceContext* c) {

      shape_inference::ShapeHandle batch_size=c->Vector(c->Dim(c->input(0),0));
      shape_inference::DimensionHandle jacobian_cols=c->Dim(c->input(0),3);
      shape_inference::DimensionHandle one=c->Dim(c->input(2),3);
      shape_inference::ShapeHandle output1_shape;
      shape_inference::ShapeHandle output2_shape;

      c->Concatenate(batch_size,c->Matrix(jacobian_cols,jacobian_cols),&output1_shape);
      c->Concatenate(batch_size,c->Matrix(jacobian_cols,one),&output2_shape);

      c->set_output(0,output1_shape);
      c->set_output(1,output2_shape);

      return Status::OK();
    });



struct Sum {
  __host__ __device__ float operator()(const float& a, const float& b) const {
    return a + b;
  }
};

__global__ void ColumnReduceSimpleKernel(const float* in,float* out, int num_planes,
                                         int num_rows, int num_cols) {

  const int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const int elems_per_plane = num_rows * num_cols;

  const int plane = gid / num_cols;
  const int col = gid % num_cols;

  if (plane >= num_planes) 
    return;

  float sum = in[plane * elems_per_plane + col]+in[plane * elems_per_plane + num_cols + col];
  for (int row = 2; row < num_rows; ++row) {
    sum = sum+in[plane * elems_per_plane + row * num_cols + col];
  }
  out[plane * num_cols + col] = sum;
}


//template <typename T, typename Op, typename OUT_T, typename IN_T>
void Launch3DYReduction(const hipStream_t& cu_stream,float* out,const float* in, int extent_x,
                        int extent_y, int extent_z) {
  int threads_per_block = 128;
  int num_blocks =(extent_x * extent_z + threads_per_block - 1) / threads_per_block;
  ColumnReduceSimpleKernel<<<num_blocks, threads_per_block, 0, cu_stream>>>(
      in, out, extent_x, extent_y, extent_z);
}

#define MAX_BUFFERS 5
#define MAX_GPUS 4

typedef perftools::gputools::DeviceMemory<float> DeviceMemoryType;
static Tensor* buffer[MAX_GPUS]={nullptr};
static std::vector<DeviceMemoryType>  memory[MAX_BUFFERS*MAX_GPUS];
static std::vector<DeviceMemoryType*> memory_ptr[MAX_BUFFERS*MAX_GPUS];


class EquationConstruction: public OpKernel 
{

private:

  int gpu_id;

public:

  explicit EquationConstruction(OpKernelConstruction* context):OpKernel(context){
    gpu_id=context->device()->tensorflow_gpu_device_info()->gpu_id;
    //std::cout<<"gpu:"<<gpu_id<<std::endl;
  }

  void Compute( OpKernelContext* context ) override {
    //std::cout<<"computing:"<<gpu_id<<std::endl;
    
    const Tensor& jacobian=context->input(0);
    const TensorShape jacobian_shape(jacobian.shape());

    const Tensor& gradient=context->input(1);
    const TensorShape gradient_shape(gradient.shape());

    const Tensor& difference=context->input(2);
    const TensorShape difference_shape(difference.shape());

    int batch_size=jacobian_shape.dim_size(0);
    int npixels   =jacobian_shape.dim_size(1);
    int n_matrix  =batch_size*npixels;

    int jacobian_rows=jacobian_shape.dim_size(2);
    int jacobian_cols=jacobian_shape.dim_size(3);

    int gradient_rows=gradient_shape.dim_size(2);
    int gradient_cols=gradient_shape.dim_size(3);

    int difference_rows=difference_shape.dim_size(2);
    int difference_cols=difference_shape.dim_size(3);


    if (buffer[gpu_id]==nullptr){

      TensorShape buffer_shape(jacobian.shape());
      buffer_shape.set_dim(2,jacobian_rows+jacobian_cols);
      PersistentTensor* newtensor= new PersistentTensor();
      context->allocate_persistent(jacobian.dtype(),buffer_shape,newtensor,&buffer[gpu_id]);


      //std::cout<<"EquationConstruction"<<buffer[gpu_id]<<std::endl;


      const float* buffer_base_ptr1 = buffer[gpu_id]->flat<float>().data();
      const float* buffer_base_ptr2 = buffer_base_ptr1+(n_matrix*jacobian_cols*jacobian_cols);

      for(int i=0;i<MAX_BUFFERS;i++){
        memory[MAX_BUFFERS*gpu_id+i].clear();
        memory[MAX_BUFFERS*gpu_id+i].reserve(n_matrix);
        memory_ptr[MAX_BUFFERS*gpu_id+i].clear();
        memory_ptr[MAX_BUFFERS*gpu_id+i].reserve(n_matrix);
      }

      for(int i=0;i<n_matrix;i++){

        memory[MAX_BUFFERS*gpu_id].push_back(AsDeviceMemory(buffer_base_ptr1+i*gradient_cols*gradient_cols));
        memory[MAX_BUFFERS*gpu_id+1].push_back(AsDeviceMemory(buffer_base_ptr2+i*jacobian_rows*jacobian_cols));
        memory[MAX_BUFFERS*gpu_id+2].push_back(AsDeviceMemory(buffer_base_ptr1+i*jacobian_cols*jacobian_cols));
        
        memory[MAX_BUFFERS*gpu_id+3].push_back(AsDeviceMemory(buffer_base_ptr2+i*gradient_cols*difference_cols));
        memory[MAX_BUFFERS*gpu_id+4].push_back(AsDeviceMemory(buffer_base_ptr1+i*jacobian_cols));

        memory_ptr[MAX_BUFFERS*gpu_id].push_back(&memory[MAX_BUFFERS*gpu_id].back());
        memory_ptr[MAX_BUFFERS*gpu_id+1].push_back(&memory[MAX_BUFFERS*gpu_id+1].back());
        memory_ptr[MAX_BUFFERS*gpu_id+2].push_back(&memory[MAX_BUFFERS*gpu_id+2].back());
        memory_ptr[MAX_BUFFERS*gpu_id+3].push_back(&memory[MAX_BUFFERS*gpu_id+3].back());
        memory_ptr[MAX_BUFFERS*gpu_id+4].push_back(&memory[MAX_BUFFERS*gpu_id+4].back());
      }
      //std::cout<<gpu_id<<" initialized"<<std::endl;
    }

    const float* jacobian_base_ptr = jacobian.flat<float>().data();
    const float* gradient_base_ptr = gradient.flat<float>().data();
    const float* difference_base_ptr=difference.flat<float>().data();

    std::vector<DeviceMemoryType>  jacobian_device_memory;
    std::vector<DeviceMemoryType>  gradient_device_memory;
    std::vector<DeviceMemoryType>  difference_device_memory;

    std::vector<DeviceMemoryType*> jacobian_ptrs;
    std::vector<DeviceMemoryType*> gradient_ptrs;
    std::vector<DeviceMemoryType*> difference_ptrs;

    jacobian_device_memory.reserve(n_matrix);
    gradient_device_memory.reserve(n_matrix);
    difference_device_memory.reserve(n_matrix);

    jacobian_ptrs.reserve(n_matrix);
    gradient_ptrs.reserve(n_matrix);
    difference_ptrs.reserve(n_matrix);

    for (int i = 0; i < n_matrix; ++i) {

      jacobian_device_memory.push_back(AsDeviceMemory(jacobian_base_ptr+i*jacobian_rows*jacobian_cols));
      gradient_device_memory.push_back(AsDeviceMemory(gradient_base_ptr+i*gradient_rows*gradient_cols));
      difference_device_memory.push_back(AsDeviceMemory(difference_base_ptr+i*difference_rows*difference_cols));

      jacobian_ptrs.push_back(&jacobian_device_memory.back());
      gradient_ptrs.push_back(&gradient_device_memory.back());
      difference_ptrs.push_back(&difference_device_memory.back());
    }

    //std::cout<<"computing0 done:"<<gpu_id<<std::endl; 
    CublasScratchAllocator scratch_allocator(context);
    bool blas_launch_status = context->op_device_context()
                                     ->stream()
                                     ->ThenBlasGemmBatchedWithScratch(
                                        perftools::gputools::blas::Transpose::kNoTranspose,
                                        perftools::gputools::blas::Transpose::kTranspose, 
                                        gradient_cols, gradient_cols,gradient_rows,static_cast<float>(1.0), 
                                        gradient_ptrs, gradient_cols, 
                                        gradient_ptrs, gradient_cols, 
                                        static_cast<float>(0.0),memory_ptr[MAX_BUFFERS*gpu_id],gradient_cols,
                                        n_matrix, &scratch_allocator).ok();

    //std::cout<<"computing1 done:"<<gpu_id<<std::endl;                           

    blas_launch_status = context->op_device_context()
                                 ->stream()
                                 ->ThenBlasGemmBatchedWithScratch(
                                    perftools::gputools::blas::Transpose::kNoTranspose,
                                    perftools::gputools::blas::Transpose::kNoTranspose, 
                                    jacobian_cols,gradient_cols,jacobian_rows,static_cast<float>(1.0), 
                                    jacobian_ptrs, jacobian_cols, 
                                    memory_ptr[MAX_BUFFERS*gpu_id],gradient_cols, 
                                    static_cast<float>(0.0),memory_ptr[MAX_BUFFERS*gpu_id+1],jacobian_cols,
                                    n_matrix, &scratch_allocator).ok();
    //std::cout<<"computing2 done:"<<gpu_id<<std::endl;                             

    blas_launch_status = context->op_device_context()
                                 ->stream()
                                 ->ThenBlasGemmBatchedWithScratch(
                                    perftools::gputools::blas::Transpose::kNoTranspose,
                                    perftools::gputools::blas::Transpose::kTranspose, 
                                    jacobian_cols,jacobian_cols,gradient_cols,static_cast<float>(1.0), 
                                    memory_ptr[MAX_BUFFERS*gpu_id+1], jacobian_cols, 
                                    jacobian_ptrs, jacobian_cols, 
                                    static_cast<float>(0.0),memory_ptr[MAX_BUFFERS*gpu_id+2],jacobian_cols,
                                    n_matrix, &scratch_allocator).ok();
    //std::cout<<"computing3 done:"<<gpu_id<<std::endl;
    
    TensorShape output_shape;
    output_shape.AddDim(batch_size);
    output_shape.AddDim(jacobian_cols);
    output_shape.AddDim(jacobian_cols);

    Tensor* output_tensor=NULL;
    OP_REQUIRES_OK(context,context->allocate_output(0,output_shape,&output_tensor));
    
    auto output = output_tensor->flat<float>();
    auto input  = buffer[gpu_id]->flat<float>();
    int nelements=jacobian_cols*jacobian_cols;

    Launch3DYReduction(GetCudaStream(context),output.data(),input.data(),batch_size,npixels,nelements);

    blas_launch_status = context->op_device_context()
                           ->stream()
                           ->ThenBlasGemmBatchedWithScratch(
                              perftools::gputools::blas::Transpose::kNoTranspose,
                              perftools::gputools::blas::Transpose::kTranspose, 
                              difference_cols,gradient_cols,gradient_rows,static_cast<float>(1.0), 
                              difference_ptrs,difference_cols, 
                              gradient_ptrs,gradient_cols, 
                              static_cast<float>(0.0),memory_ptr[MAX_BUFFERS*gpu_id+3],difference_cols,
                              n_matrix, &scratch_allocator).ok();

    blas_launch_status = context->op_device_context()
                       ->stream()
                       ->ThenBlasGemmBatchedWithScratch(
                          perftools::gputools::blas::Transpose::kNoTranspose,
                          perftools::gputools::blas::Transpose::kTranspose, 
                          difference_cols,jacobian_cols,jacobian_rows,static_cast<float>(1.0), 
                          memory_ptr[MAX_BUFFERS*gpu_id+3],difference_cols, 
                          jacobian_ptrs, jacobian_cols, 
                          static_cast<float>(0.0),memory_ptr[MAX_BUFFERS*gpu_id+4],difference_cols,
                          n_matrix, &scratch_allocator).ok();

    TensorShape output2_shape;
    output2_shape.AddDim(batch_size);
    output2_shape.AddDim(jacobian_cols);
    output2_shape.AddDim(difference_cols);

    Tensor* output2_tensor=NULL;
    OP_REQUIRES_OK(context,context->allocate_output(1,output2_shape,&output2_tensor));

    auto output2 = output2_tensor->flat<float>();
    nelements=jacobian_cols*difference_cols;
    Launch3DYReduction(GetCudaStream(context),output2.data(),input.data(),batch_size,npixels,nelements);
  }
};
REGISTER_KERNEL_BUILDER(Name("EquationConstruction").Device(DEVICE_GPU),EquationConstruction);


REGISTER_OP("EquationConstructionGrad")
     .Input("jacobian:float")
     .Input("gradient:float")
     .Input("difference:float")
     .Input("left_grad:float")
     .Input("right_grad:float")
     .Output("jacobian_grad:float")
     .Output("gradient_grad:float")
     .Output("difference_grad:float");
     // .Output("test_output:float");

#define GRAD_MAX_BUFFERS 4 
enum{
  GRAD_0,
  GRAD_1,
  JACOBIAN_GRAD,
  JACOBIAN_GRAD_GRAD
};
static Tensor* grad_buffer[MAX_GPUS]={nullptr};
static std::vector<DeviceMemoryType>  grad_memory[GRAD_MAX_BUFFERS*MAX_GPUS];
static std::vector<DeviceMemoryType*> grad_memory_ptr[GRAD_MAX_BUFFERS*MAX_GPUS];

__global__ void tile_kernel(const float* in,float* out, 
                           int num_planes, int num_rows, int num_cols) {

  const int gid = threadIdx.x + blockIdx.x * blockDim.x;
  const int elems_per_plane = num_rows * num_cols;

  const int plane = gid / num_rows;
  const int row   = gid % num_rows;

  if (plane >= num_planes) 
    return;

  for (int col=0;col<num_cols; ++col){
    out[plane * elems_per_plane + row * num_cols + col]=in[plane*num_cols+col];
  }
}

void tile_gpu(const hipStream_t& stream,float* out,const float* in,const int batch_size,const int npixels,const int nelements){
  int threads_per_block = 128;
  int num_blocks =(batch_size * npixels+ threads_per_block - 1) / threads_per_block;
  tile_kernel<<<num_blocks,threads_per_block,0,stream>>>(in,out,batch_size,npixels,nelements);
}

class EquationConstructionGrad: public OpKernel 
{
private:

  int gpu_id;

public:

  explicit EquationConstructionGrad(OpKernelConstruction* context):OpKernel(context){
    gpu_id=context->device()->tensorflow_gpu_device_info()->gpu_id;
  }
  void Compute( OpKernelContext* context ) override {

    const Tensor& jacobian=context->input(0);
    const TensorShape jacobian_shape(jacobian.shape());

    const Tensor& gradient=context->input(1);
    const TensorShape gradient_shape(gradient.shape());

    const Tensor& difference=context->input(2);
    const TensorShape difference_shape(difference.shape());

    const Tensor& grad0=context->input(3);
    const Tensor& grad1=context->input(4);

    Tensor* jacobian_grad_tensor=NULL;
    OP_REQUIRES_OK(context,context->allocate_output(0,jacobian_shape,&jacobian_grad_tensor));

    Tensor* gradient_grad_tensor=NULL;
    OP_REQUIRES_OK(context,context->allocate_output(1,gradient_shape,&gradient_grad_tensor));

    Tensor* difference_grad_tensor=NULL;
    OP_REQUIRES_OK(context,context->allocate_output(2,difference_shape,&difference_grad_tensor));


    int batch_size=jacobian_shape.dim_size(0);
    int npixels   =jacobian_shape.dim_size(1);
    int n_matrix  =batch_size*npixels;

    int jacobian_rows=jacobian_shape.dim_size(2);
    int jacobian_cols=jacobian_shape.dim_size(3);

    int gradient_rows=gradient_shape.dim_size(2);
    int gradient_cols=gradient_shape.dim_size(3);

    int difference_rows=difference_shape.dim_size(2);
    int difference_cols=difference_shape.dim_size(3);


    // std::cout<<buffer[gpu_id]<<std::endl;
    float* tiled_grad0_ptr=buffer[gpu_id]->flat<float>().data();
    float* tiled_grad1_ptr=tiled_grad0_ptr+(n_matrix*jacobian_cols*jacobian_cols);
    

    if (grad_buffer[gpu_id]==nullptr){

      TensorShape buffer_shape(jacobian.shape());
      buffer_shape.set_dim(2,2*gradient_rows);
      PersistentTensor* newtensor= new PersistentTensor();
      context->allocate_persistent(jacobian.dtype(),buffer_shape,newtensor,&grad_buffer[gpu_id]);
        
      float* buffer_base_ptr1 = grad_buffer[gpu_id]->flat<float>().data();
      float* buffer_base_ptr2 = buffer_base_ptr1+(n_matrix*gradient_rows*jacobian_cols);

      for(int i=0;i<GRAD_MAX_BUFFERS;i++){
        grad_memory[GRAD_MAX_BUFFERS*gpu_id+i].clear();
        grad_memory[GRAD_MAX_BUFFERS*gpu_id+i].reserve(n_matrix);
        grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+i].clear();
        grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+i].reserve(n_matrix);
      }

      for(int i=0;i<n_matrix;i++){

        grad_memory[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD].push_back(AsDeviceMemory(buffer_base_ptr2+i*gradient_rows*jacobian_cols));
        grad_memory[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD].push_back(AsDeviceMemory(buffer_base_ptr1+i*gradient_rows*jacobian_cols));

        grad_memory[GRAD_MAX_BUFFERS*gpu_id+GRAD_0].push_back(AsDeviceMemory(tiled_grad0_ptr+i*jacobian_cols*jacobian_cols));
        grad_memory[GRAD_MAX_BUFFERS*gpu_id+GRAD_1].push_back(AsDeviceMemory(tiled_grad1_ptr+i*difference_cols*jacobian_cols));

        grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD].push_back(&grad_memory[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD].back());
        grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD].push_back(&grad_memory[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD].back());

        grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+GRAD_0].push_back(&grad_memory[GRAD_MAX_BUFFERS*gpu_id+GRAD_0].back());
        grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+GRAD_1].push_back(&grad_memory[GRAD_MAX_BUFFERS*gpu_id+GRAD_1].back());

      }
    }

    const float* jacobian_base_ptr = jacobian.flat<float>().data();
    const float* gradient_base_ptr = gradient.flat<float>().data();
    const float* difference_base_ptr=difference.flat<float>().data();

    const float* jacobian_grad_base_ptr=jacobian_grad_tensor->flat<float>().data();
    const float* gradient_grad_base_ptr=gradient_grad_tensor->flat<float>().data();
    const float* difference_grad_base_ptr=difference_grad_tensor->flat<float>().data();

    std::vector<DeviceMemoryType>  jacobian_device_memory;
    std::vector<DeviceMemoryType>  gradient_device_memory;
    std::vector<DeviceMemoryType>  difference_device_memory;

    std::vector<DeviceMemoryType>  jacobian_grad_device_memory;
    std::vector<DeviceMemoryType>  gradient_grad_device_memory;
    std::vector<DeviceMemoryType>  difference_grad_device_memory;

    std::vector<DeviceMemoryType*> jacobian_ptrs;
    std::vector<DeviceMemoryType*> gradient_ptrs;
    std::vector<DeviceMemoryType*> difference_ptrs;

    std::vector<DeviceMemoryType*> jacobian_grad_ptrs;
    std::vector<DeviceMemoryType*> gradient_grad_ptrs;
    std::vector<DeviceMemoryType*> difference_grad_ptrs;

    jacobian_device_memory.reserve(n_matrix);
    gradient_device_memory.reserve(n_matrix);
    difference_device_memory.reserve(n_matrix);

    jacobian_grad_device_memory.reserve(n_matrix);
    gradient_grad_device_memory.reserve(n_matrix);
    difference_grad_device_memory.reserve(n_matrix);

    jacobian_ptrs.reserve(n_matrix);
    gradient_ptrs.reserve(n_matrix);
    difference_ptrs.reserve(n_matrix);

    jacobian_grad_ptrs.reserve(n_matrix);
    gradient_grad_ptrs.reserve(n_matrix);
    difference_grad_ptrs.reserve(n_matrix);

    for (int i = 0; i < n_matrix; ++i) {

      jacobian_device_memory.push_back(AsDeviceMemory(jacobian_base_ptr+i*jacobian_rows*jacobian_cols));
      gradient_device_memory.push_back(AsDeviceMemory(gradient_base_ptr+i*gradient_rows*gradient_cols));
      difference_device_memory.push_back(AsDeviceMemory(difference_base_ptr+i*difference_rows*difference_cols));

      jacobian_ptrs.push_back(&jacobian_device_memory.back());
      gradient_ptrs.push_back(&gradient_device_memory.back());
      difference_ptrs.push_back(&difference_device_memory.back());


      jacobian_grad_device_memory.push_back(AsDeviceMemory(jacobian_grad_base_ptr+i*jacobian_rows*jacobian_cols));
      gradient_grad_device_memory.push_back(AsDeviceMemory(gradient_grad_base_ptr+i*gradient_rows*gradient_cols));
      difference_grad_device_memory.push_back(AsDeviceMemory(difference_grad_base_ptr+i*difference_rows*difference_cols));

      jacobian_grad_ptrs.push_back(&jacobian_grad_device_memory.back());
      gradient_grad_ptrs.push_back(&gradient_grad_device_memory.back());
      difference_grad_ptrs.push_back(&difference_grad_device_memory.back());
    }
    
    int grad0_nelements=jacobian_cols*jacobian_cols;
    tile_gpu(GetCudaStream(context),tiled_grad0_ptr,grad0.flat<float>().data(),batch_size,npixels,grad0_nelements);

    int grad1_nelements=jacobian_cols*difference_cols;
    tile_gpu(GetCudaStream(context),tiled_grad1_ptr,grad1.flat<float>().data(),batch_size,npixels,grad1_nelements);


    



    CublasScratchAllocator scratch_allocator(context);
    bool blas_launch_status = context->op_device_context()
                                                ->stream()
                                                ->ThenBlasGemmBatchedWithScratch(
                                                   perftools::gputools::blas::Transpose::kNoTranspose,
                                                   perftools::gputools::blas::Transpose::kNoTranspose, 
                                                   jacobian_cols,gradient_rows,jacobian_rows,static_cast<float>(1.0), 
                                                   jacobian_ptrs, jacobian_cols, 
                                                   gradient_ptrs, gradient_cols, 
                                                   static_cast<float>(0.0),grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD],jacobian_cols,
                                                   n_matrix, &scratch_allocator).ok();

    blas_launch_status = context->op_device_context()
                                           ->stream()
                                           ->ThenBlasGemmBatchedWithScratch(
                                              perftools::gputools::blas::Transpose::kNoTranspose,
                                              perftools::gputools::blas::Transpose::kNoTranspose, 
                                              difference_cols,gradient_rows,jacobian_cols,static_cast<float>(1.0), 
                                              grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+GRAD_1],difference_cols, 
                                              grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD],jacobian_cols, 
                                              static_cast<float>(0.0),difference_grad_ptrs,difference_cols,
                                              n_matrix, &scratch_allocator).ok();

    
    blas_launch_status = context->op_device_context()
                                           ->stream()
                                           ->ThenBlasGemmBatchedWithScratch(
                                              perftools::gputools::blas::Transpose::kNoTranspose,
                                              perftools::gputools::blas::Transpose::kNoTranspose, 
                                              jacobian_cols,gradient_rows,jacobian_cols,static_cast<float>(2.0), 
                                              grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+GRAD_0],jacobian_cols, 
                                              grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD],jacobian_cols, 
                                              static_cast<float>(0.0),grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD],jacobian_cols,
                                              n_matrix, &scratch_allocator).ok();

    blas_launch_status = context->op_device_context()
                                           ->stream()
                                           ->ThenBlasGemmBatchedWithScratch(
                                              perftools::gputools::blas::Transpose::kTranspose,
                                              perftools::gputools::blas::Transpose::kNoTranspose, 
                                              jacobian_cols,gradient_rows,difference_cols,static_cast<float>(1.0), 
                                              grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+GRAD_1],difference_cols, 
                                              difference_ptrs,difference_cols, 
                                              static_cast<float>(1.0),grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD],jacobian_cols,
                                              n_matrix, &scratch_allocator).ok();

    blas_launch_status = context->op_device_context()
                                       ->stream()
                                       ->ThenBlasGemmBatchedWithScratch(
                                          perftools::gputools::blas::Transpose::kNoTranspose,
                                          perftools::gputools::blas::Transpose::kTranspose, 
                                          jacobian_cols,gradient_cols,gradient_rows,static_cast<float>(1.0), 
                                          grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD],jacobian_cols, 
                                          gradient_ptrs,gradient_cols, 
                                          static_cast<float>(0.0),jacobian_grad_ptrs,jacobian_cols,
                                          n_matrix, &scratch_allocator).ok(); 

    blas_launch_status = context->op_device_context()
                                       ->stream()
                                       ->ThenBlasGemmBatchedWithScratch(
                                          perftools::gputools::blas::Transpose::kTranspose,
                                          perftools::gputools::blas::Transpose::kNoTranspose, 
                                          gradient_cols,gradient_rows,jacobian_cols,static_cast<float>(1.0), 
                                          jacobian_ptrs,jacobian_cols, 
                                          grad_memory_ptr[GRAD_MAX_BUFFERS*gpu_id+JACOBIAN_GRAD_GRAD],jacobian_cols, 
                                          static_cast<float>(0.0),gradient_grad_ptrs,gradient_cols,
                                          n_matrix, &scratch_allocator).ok();                                                                                                            

  }
};
REGISTER_KERNEL_BUILDER(Name("EquationConstructionGrad").Device(DEVICE_GPU),EquationConstructionGrad);

